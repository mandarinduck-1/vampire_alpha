#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
//
// This source file is part of the VAMPIRE open source package under the
// GNU GPL (version 2) licence (see licence file for details).
//
// (c) R F L Evans 2015. All rights reserved.
// Reviewed: Andrea Meo 2022
//
//-----------------------------------------------------------------------------

// C++ standard library headers

#include <vector>

// Vampire headers
#include "anisotropy.hpp"
#include "atoms.hpp"
#include "cuda.hpp"
#include "errors.hpp"
#include "dipole.hpp"
#include "hamr.hpp"
#include "gpu.hpp"
#include "random.hpp"
#include "stats.hpp"
#include "typedefs.hpp"
#include "vio.hpp"

// Local cuda headers

#include "cuda_utils.hpp"
#include "data.hpp"
#include "internal.hpp"

#include "exchange_fields.hpp"
#include "statistics.hpp"

#include "monte_carlo.hpp"

#ifdef CUDA
namespace cu = ::vcuda::internal;
#endif

namespace vcuda
{

   //-------------------------------------------------------------------------------
   // Function to initialize GPU data
   //-------------------------------------------------------------------------------
   bool initialize(bool cpu_stats)
   {

#ifdef CUDA

      std::cout << "CUDA has been enabled in ";
      zlog << zTs() << "CUDA has been enabled in ";
#ifdef CUDA_DP
      std::cout << "double precision mode" << std::endl;
      zlog << "double precision mode" << std::endl;
#else
      std::cout << "single precision mode" << std::endl;
      zlog << "single precision mode" << std::endl;
#endif

      // set internal cpu statistics flag
      vcuda::internal::stats::use_cpu = cpu_stats;

      // Check if there is a compatible devices
      int n_devices;
      hipError_t error = hipGetDeviceCount(&n_devices);

      if (error == hipErrorNoDevice)
      {
         std::cout << "Error: CUDA is enabled but no CUDA compatible devices are available." << std::endl;
         zlog << zTs() << "Error: CUDA is enabled but no CUDA compatible devices are available." << std::endl;
         ::err::vexit();
      }
      else if (error == hipErrorInsufficientDriver)
      {
         std::cout << "Error: CUDA is enabled but no CUDA drivers are incompatible. Please update drivers." << std::endl;
         zlog << zTs() << "Error: CUDA is enabled but no CUDA drivers are incompatible. Please update drivers." << std::endl;
         ::err::vexit();
      }
      else if (error != hipSuccess)
      {
         std::cout << "Error: CUDA is enabled but error querying devices." << std::endl;
         zlog << zTs() << "Error: CUDA is enabled but error querying devices." << std::endl;
         ::err::vexit();
      }

      // Set cuda device if specified by user
      if (gpu::device != -1 && gpu::device < n_devices)
      {
         zlog << zTs() << "Setting CUDA device to " << gpu::device << std::endl;
         hipError_t error = hipSetDevice(gpu::device);
         if (error == hipErrorSetOnActiveProcess)
         {
            std::cerr << "Error: CUDA is unable to set active process to device " << gpu::device << std::endl;
            zlog << zTs() << "Error: CUDA is unable to set active process to device " << gpu::device << std::endl;
            ::err::vexit();
         }
         else if (error == hipErrorInvalidDevice)
         {
            std::cerr << "Error: CUDA is requesting device " << gpu::device << " which is an invalid device." << std::endl;
            zlog << zTs() << "Error: CUDA is requesting device " << gpu::device << " which is an invalid device." << std::endl;
            ::err::vexit();
         }
         else if (error != hipSuccess)
         {
            std::cerr << "Error: CUDA unable to set device to " << gpu::device << std::endl;
            zlog << zTs() << "Error: CUDA unable to set device to " << gpu::device << std::endl;
            ::err::vexit();
         }
      }

      bool success = true;

      /*
       * Set the block_size according to the number of atoms
       */

      size_t _grid_size = ((::atoms::num_atoms / 2) / cu::block_size) + 1;

      // std::cerr << "Natoms = " << ::atoms::num_atoms << "\n";
      // std::cerr << "Block size = " << cu::block_size << "\n";
      // std::cerr << "grid1 = " << _grid_size << "\n";
      // std::cerr << "grid2 = " << cu::grid_size << "\n";

      // I dont think this does anything
      // if (_grid_size < cu::grid_size)
      //   cu::grid_size = _grid_size;

      cu::grid_size = _grid_size;
      // std::cerr << "grid2 = " << cu::grid_size << std::endl;

      zlog << zTs() << "Using cuda version with block size " << cu::block_size << " and grid size " << cu::grid_size << std::endl;

      success = success && cu::__initialize_atoms();
      success = success && cu::__initialize_fields();
      success = success && cu::__initialize_cells();
      success = success && cu::__initialize_materials();
      success = success && cu::__initialize_topology();
      success = success && cu::__initialize_curand();
      success = success && cu::__initialize_stats();
      success = success && cu::__initialize_hamr();

      // Set up the exchange fields
      if (cu::exchange::initialise_exchange() != EXIT_SUCCESS)
      {
         std::cerr << "Failed to initialise exchange" << std::endl;
         success = false;
      }

      cu::mc::initialise();

      // Successful initialization
      return success;

#else
      // Default (initializtion failed)
      return false;
#endif
   }

   bool initialize_dipole()
   {
#ifdef CUDA

      bool success = true;

      // Initialise dipole
      if (cu::__initialize_dipole() != true)
      {
         std::cerr << "Failed to initialise dipole" << std::endl;
         success = false;
      }

      // Successful initialization
      return success;
#else
      // Default (initializtion failed)
      return false;
#endif
   }

#ifdef CUDA

   namespace internal
   {

      bool __initialize_atoms()
      {
         /*
          * Allocate memory in the device and transfer the
          * spins of the atoms.
          */
         /*
         cu::atoms::x_spin_array.resize(::atoms::num_atoms);
         cu::atoms::y_spin_array.resize(::atoms::num_atoms);
         cu::atoms::z_spin_array.resize(::atoms::num_atoms);
         */

         size_t num_bytes = ::atoms::num_atoms * sizeof(cu_real_t);

         hipMalloc((void **)&cu::atoms::d_spin, num_bytes * 3);

         cu::atoms::d_x_spin = cu::atoms::d_spin;
         cu::atoms::d_y_spin = cu::atoms::d_spin + ::atoms::num_atoms;
         cu::atoms::d_z_spin = cu::atoms::d_spin + 2 * ::atoms::num_atoms;

         /*
         hipMalloc((void**)&cu::atoms::d_x_spin, num_bytes);
              hipMalloc((void**)&cu::atoms::d_y_spin, num_bytes);
              hipMalloc((void**)&cu::atoms::d_z_spin, num_bytes);
              */
         /* Need to be careful here
              The device code can use SP or DP,
              but the host code seems to rely exclusively on DP */

         std::vector<cu_real_t> tmp_buffer;
         tmp_buffer.resize(::atoms::num_atoms);

         hipHostMalloc((void **)&cu::h_x_spin_transfer_buffer, num_bytes);
         hipHostMalloc((void **)&cu::h_y_spin_transfer_buffer, num_bytes);
         hipHostMalloc((void **)&cu::h_z_spin_transfer_buffer, num_bytes);

         std::copy(::atoms::x_spin_array.begin(), ::atoms::x_spin_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::atoms::d_x_spin, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::atoms::y_spin_array.begin(), ::atoms::y_spin_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::atoms::d_y_spin, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::atoms::z_spin_array.begin(), ::atoms::z_spin_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::atoms::d_z_spin, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         /*
          * Allocate memory in the device and transfer the
          * coordinates of the atoms.
          */

         hipMalloc((void **)&cu::atoms::d_x_coord, num_bytes);
         hipMalloc((void **)&cu::atoms::d_y_coord, num_bytes);
         hipMalloc((void **)&cu::atoms::d_z_coord, num_bytes);

         std::copy(::atoms::x_coord_array.begin(), ::atoms::x_coord_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::atoms::d_x_coord, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::atoms::y_coord_array.begin(), ::atoms::y_coord_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::atoms::d_y_coord, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::atoms::z_coord_array.begin(), ::atoms::z_coord_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::atoms::d_z_coord, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         /*
          * Allocate memory and send information about the types of
          * atoms
          */

         // cu::atoms::type_array.resize(::atoms::num_atoms);

         hipMalloc((void **)&cu::atoms::d_materials, ::atoms::num_atoms * sizeof(int));
         hipMemcpy(cu::atoms::d_materials, ::atoms::type_array.data(), ::atoms::num_atoms * sizeof(int), hipMemcpyHostToDevice);

         /*
          * Allocate memory and pass the cell information
          */

         hipMalloc((void **)&cu::atoms::d_cells, ::atoms::num_atoms * sizeof(int));
         //         hipMemcpy(cu::atoms::d_cells, ::atoms::cell_array.data(), ::atoms::num_atoms * sizeof(int), hipMemcpyHostToDevice);
         hipMemcpy(cu::atoms::d_cells, ::cells::atom_cell_id_array.data(), ::atoms::num_atoms * sizeof(int), hipMemcpyHostToDevice);

         /*
          * Allocate the memory for the unrolled spin norm array
          */

         // This is actually used in Thrust algorithms in statistics.cu
         // Leave it for now
         // cu::atoms::spin_norm_array.resize(::atoms::num_atoms);

         // thrust::copy(
         //       ::atoms::m_spin_array.begin(),
         //       ::atoms::m_spin_array.end(),
         //       cu::atoms::spin_norm_array.begin()
         //       );

         return true;
      }

      bool __initialize_fields()
      {
         /*
          * Allocate memory in the device and transfer the
          * total spin field in each atom.
          */

         size_t num_bytes = ::atoms::num_atoms * sizeof(cu_real_t);
         std::vector<cu_real_t> tmp_buffer;
         tmp_buffer.resize(::atoms::num_atoms);

         // hipMalloc((void**)&cu::d_x_spin_field, num_bytes);
         // hipMalloc((void**)&cu::d_y_spin_field, num_bytes);
         // hipMalloc((void**)&cu::d_z_spin_field, num_bytes);

         hipMalloc((void **)&cu::d_spin_field, num_bytes * 3);

         cu::d_x_spin_field = cu::d_spin_field;
         cu::d_y_spin_field = cu::d_spin_field + ::atoms::num_atoms;
         cu::d_z_spin_field = cu::d_spin_field + 2 * ::atoms::num_atoms;

         std::copy(::atoms::x_total_spin_field_array.begin(), ::atoms::x_total_spin_field_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_x_spin_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::atoms::y_total_spin_field_array.begin(), ::atoms::y_total_spin_field_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_y_spin_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::atoms::z_total_spin_field_array.begin(), ::atoms::z_total_spin_field_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_z_spin_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         /*
          * Allocate memory in the device and transfer the
          * total external field in each atom.
          */

         hipMalloc((void **)&cu::d_x_external_field, num_bytes);
         hipMalloc((void **)&cu::d_y_external_field, num_bytes);
         hipMalloc((void **)&cu::d_z_external_field, num_bytes);

         hipMemset(cu::d_x_external_field, 0.0, num_bytes);
         hipMemset(cu::d_y_external_field, 0.0, num_bytes);
         hipMemset(cu::d_z_external_field, 0.0, num_bytes);

         /* // It should not be necessary to copy the external field at initialisation
         std::copy(::atoms::x_total_external_field_array.begin(), ::atoms::x_total_external_field_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_x_external_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::atoms::y_total_external_field_array.begin(), ::atoms::y_total_external_field_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_y_external_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::atoms::z_total_external_field_array.begin(), ::atoms::z_total_external_field_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_z_external_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice); */

         /*
          * Allocate memory and transfer any existing
          * initial data for the dipolar field
          */

         hipMalloc((void **)&cu::d_x_dip_field, num_bytes);
         hipMalloc((void **)&cu::d_y_dip_field, num_bytes);
         hipMalloc((void **)&cu::d_z_dip_field, num_bytes);

         std::copy(::dipole::atom_dipolar_field_array_x.begin(), ::dipole::atom_dipolar_field_array_x.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_x_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::dipole::atom_dipolar_field_array_y.begin(), ::dipole::atom_dipolar_field_array_y.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_y_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::dipole::atom_dipolar_field_array_z.begin(), ::dipole::atom_dipolar_field_array_z.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_z_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         hipMalloc((void **)&cu::d_x_mu0H_dip_field, num_bytes);
         hipMalloc((void **)&cu::d_y_mu0H_dip_field, num_bytes);
         hipMalloc((void **)&cu::d_z_mu0H_dip_field, num_bytes);

         std::copy(::dipole::atom_mu0demag_field_array_x.begin(), ::dipole::atom_mu0demag_field_array_x.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_x_mu0H_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::dipole::atom_mu0demag_field_array_y.begin(), ::dipole::atom_mu0demag_field_array_y.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_y_mu0H_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::dipole::atom_mu0demag_field_array_z.begin(), ::dipole::atom_mu0demag_field_array_z.end(), tmp_buffer.begin());
         hipMemcpy(cu::d_z_mu0H_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         // ======唐愈涵加的目的是实现局部场======
         // 分配局部场设备内存
         hipMalloc((void **)&cu::local_field_x, num_bytes);
         hipMalloc((void **)&cu::local_field_y, num_bytes);
         hipMalloc((void **)&cu::local_field_z, num_bytes);

         // 从主机复制局部场数据到设备
         std::copy(::cells::local_field_array_x.begin(), ::cells::local_field_array_x.end(), tmp_buffer.begin());
         hipMemcpy(cu::local_field_x, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::cells::local_field_array_y.begin(), ::cells::local_field_array_y.end(), tmp_buffer.begin());
         hipMemcpy(cu::local_field_y, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::cells::local_field_array_z.begin(), ::cells::local_field_array_z.end(), tmp_buffer.begin());
         hipMemcpy(cu::local_field_z, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         // ============================

         return true;
      }

      bool __initialize_cells()
      {
         /*
          * Allocate memory and initialize coordinates
          */

         size_t num_bytes = ::cells::num_cells * sizeof(cu_real_t);
         std::vector<cu_real_t> tmp_buffer;
         tmp_buffer.resize(::cells::num_cells);

         hipMalloc((void **)&cu::cells::d_x_coord, num_bytes);
         hipMalloc((void **)&cu::cells::d_y_coord, num_bytes);
         hipMalloc((void **)&cu::cells::d_z_coord, num_bytes);

         /*cu::cells::x_coord_array.resize(::cells::num_cells);
         cu::cells::y_coord_array.resize(::cells::num_cells);
         cu::cells::z_coord_array.resize(::cells::num_cells);
         */
         // unroll 4N array to N
         std::vector<cu_real_t> pos(::cells::num_cells, 0.0);
         for (int cell = 0; cell < pos.size(); cell++)
            pos[cell] = ::cells::pos_and_mom_array[4 * cell + 0]; // x

         hipMemcpy(cu::cells::d_x_coord, pos.data(), ::cells::num_cells * sizeof(cu_real_t), hipMemcpyHostToDevice);

         // unroll 4N array to N
         for (int cell = 0; cell < pos.size(); cell++)
            pos[cell] = ::cells::pos_and_mom_array[4 * cell + 1]; // y

         hipMemcpy(cu::cells::d_y_coord, pos.data(), ::cells::num_cells * sizeof(cu_real_t), hipMemcpyHostToDevice);

         // unroll 4N array to N
         for (int cell = 0; cell < pos.size(); cell++)
            pos[cell] = ::cells::pos_and_mom_array[4 * cell + 2]; // z

         hipMemcpy(cu::cells::d_z_coord, pos.data(), ::cells::num_cells * sizeof(cu_real_t), hipMemcpyHostToDevice);

         //-----------------------------------------------------
         // Allocate memory and initialize cell magnetization
         //-----------------------------------------------------

         hipMalloc((void **)&cu::cells::d_x_mag, num_bytes);
         hipMalloc((void **)&cu::cells::d_y_mag, num_bytes);
         hipMalloc((void **)&cu::cells::d_z_mag, num_bytes);

         std::copy(::cells::mag_array_x.begin(), ::cells::mag_array_x.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_x_mag, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::cells::mag_array_y.begin(), ::cells::mag_array_y.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_y_mag, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::cells::mag_array_z.begin(), ::cells::mag_array_z.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_z_mag, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         //----------------------------------------------
         // Allocate memory and initialize cell fields
         //----------------------------------------------

         /*
          * Copy volume and number of atoms for each cell
          */
         hipMalloc((void **)&cu::cells::d_volume, num_bytes);
         std::copy(::cells::volume_array.begin(), ::cells::volume_array.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_volume, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         hipMalloc((void **)&cu::cells::d_num_atoms, ::cells::num_cells * sizeof(int));
         hipMemcpy(cu::cells::d_num_atoms, ::cells::num_atoms_in_cell.data(), ::cells::num_cells * sizeof(int), hipMemcpyHostToDevice);

         hipMalloc((void **)&cu::cells::d_cell_id_array, ::cells::cell_id_array.size() * sizeof(int));
         hipMemcpy(cu::cells::d_cell_id_array, ::cells::cell_id_array.data(), ::cells::cell_id_array.size() * sizeof(int), hipMemcpyHostToDevice);

         return true;
      }

      bool __initialize_dipole()
      {

         // Initialise and copy dipolar fields for cells.
         // It's done here because otherwise these objects are not yet initialised on the host when initialise_dipole() is called
         size_t num_bytes = ::cells::num_cells * sizeof(cu_real_t);
         std::vector<cu_real_t> tmp_buffer;
         tmp_buffer.resize(::cells::num_cells);

         hipMalloc((void **)&cu::cells::d_x_cell_field, num_bytes);
         hipMalloc((void **)&cu::cells::d_y_cell_field, num_bytes);
         hipMalloc((void **)&cu::cells::d_z_cell_field, num_bytes);

         std::copy(::dipole::cells_field_array_x.begin(), ::dipole::cells_field_array_x.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_x_cell_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::dipole::cells_field_array_y.begin(), ::dipole::cells_field_array_y.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_y_cell_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::dipole::cells_field_array_z.begin(), ::dipole::cells_field_array_z.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_z_cell_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         check_cuda_errors(__FILE__, __LINE__);

         hipMalloc((void **)&cu::cells::d_x_cell_mu0H_field, num_bytes);
         hipMalloc((void **)&cu::cells::d_y_cell_mu0H_field, num_bytes);
         hipMalloc((void **)&cu::cells::d_z_cell_mu0H_field, num_bytes);

         std::copy(::dipole::cells_mu0Hd_field_array_x.begin(), ::dipole::cells_mu0Hd_field_array_x.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_x_cell_mu0H_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::dipole::cells_mu0Hd_field_array_y.begin(), ::dipole::cells_mu0Hd_field_array_y.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_y_cell_mu0H_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
         std::copy(::dipole::cells_mu0Hd_field_array_z.begin(), ::dipole::cells_mu0Hd_field_array_z.end(), tmp_buffer.begin());
         hipMemcpy(cu::cells::d_z_cell_mu0H_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

         check_cuda_errors(__FILE__, __LINE__);

         std::vector<int> num_atoms_in_cell = ::dipole::get_num_atoms_in_cell_array();
         hipMalloc((void **)&cu::cells::d_num_atoms_in_cell, num_atoms_in_cell.size() * sizeof(int));
         hipMemcpy(cu::cells::d_num_atoms_in_cell, num_atoms_in_cell.data(), num_atoms_in_cell.size() * sizeof(int), hipMemcpyHostToDevice);

         check_cuda_errors(__FILE__, __LINE__);

         // Initialise and copy dipolar tensor

         // Copy into <cu_real_t> vectors to avoid having to perform later std::copy()
         num_bytes = ::cells::num_cells * ::cells::num_local_cells * sizeof(cu_real_t);

         cu_real_t precision; // dummy variable to allow overflow
         std::vector<cu_real_t> tensor_xx = ::dipole::unroll_tensor(1, precision);
         std::vector<cu_real_t> tensor_xy = ::dipole::unroll_tensor(2, precision);
         std::vector<cu_real_t> tensor_xz = ::dipole::unroll_tensor(3, precision);
         std::vector<cu_real_t> tensor_yy = ::dipole::unroll_tensor(4, precision);
         std::vector<cu_real_t> tensor_yz = ::dipole::unroll_tensor(5, precision);
         std::vector<cu_real_t> tensor_zz = ::dipole::unroll_tensor(6, precision);

         hipMalloc((void **)&cu::cells::d_tensor_xx, num_bytes);
         hipMalloc((void **)&cu::cells::d_tensor_xy, num_bytes);
         hipMalloc((void **)&cu::cells::d_tensor_xz, num_bytes);
         hipMalloc((void **)&cu::cells::d_tensor_yy, num_bytes);
         hipMalloc((void **)&cu::cells::d_tensor_yz, num_bytes);
         hipMalloc((void **)&cu::cells::d_tensor_zz, num_bytes);

         hipMemcpy(cu::cells::d_tensor_xx, tensor_xx.data(), num_bytes, hipMemcpyHostToDevice);
         hipMemcpy(cu::cells::d_tensor_xy, tensor_xy.data(), num_bytes, hipMemcpyHostToDevice);
         hipMemcpy(cu::cells::d_tensor_xz, tensor_xz.data(), num_bytes, hipMemcpyHostToDevice);
         hipMemcpy(cu::cells::d_tensor_yy, tensor_yy.data(), num_bytes, hipMemcpyHostToDevice);
         hipMemcpy(cu::cells::d_tensor_yz, tensor_yz.data(), num_bytes, hipMemcpyHostToDevice);
         hipMemcpy(cu::cells::d_tensor_zz, tensor_zz.data(), num_bytes, hipMemcpyHostToDevice);

         check_cuda_errors(__FILE__, __LINE__);

         check_device_memory(__FILE__, __LINE__);

         return true;
      }

      // Function to initialise hamr parameters
      bool __initialize_hamr()
      {
         // Initialise HAMR variables if hamr on cpu has been initialised
         bool initialised = ::hamr::get_initialisation_state();
         if (initialised == true)
         {
            zlog << zTs() << "Importing HAMR parameters on GPU ..." << std::endl;
            cu::hamr::d_head_position_x = ::hamr::get_head_position_x();
            cu::hamr::d_head_position_y = ::hamr::get_head_position_y();
            cu::hamr::d_laser_sigma_x = ::hamr::get_laser_sigma_x();
            cu::hamr::d_laser_sigma_y = ::hamr::get_laser_sigma_y();
            cu::hamr::d_H_bounds_x = ::hamr::get_field_bounds_x();
            cu::hamr::d_H_bounds_y = ::hamr::get_field_bounds_y();
            cu::hamr::d_NPS = ::hamr::get_NPS();
         }

         return true;
      }

      bool __initialize_materials()
      {

         /*
          * Serialize material data
          */
         size_t num_mats = ::mp::num_materials;
         std::vector<material_parameters_t> _materials(num_mats);
         for (size_t i = 0; i < num_mats; i++)
         {
            double mu_s_SI = ::mp::material[i].mu_s_SI;

            double ku2 = ::anisotropy::get_ku2(i); // second order uniaxial anisotropy constant (Ku1)
            double ku4 = ::anisotropy::get_ku4(i); // fourth order uniaxial anisotropy constant (Ku2)
            double ku6 = ::anisotropy::get_ku6(i); // sixth order uniaxial anisotropy constant  (Ku3)
            double kc4 = ::anisotropy::get_kc4(i); // fourth order cubic anisotropy constant (Kc1)
            // double kc6 = ::anisotropy::get_kc6(i); // sixth order cubic anisotropy constant (Kc2)

            std::vector<double> ku_vector = ::anisotropy::get_ku_vector(i); // unit vector defining axis for uniaxial anisotropy

            _materials[i].alpha = ::mp::material[i].alpha;
            _materials[i].gamma_rel = ::mp::material[i].gamma_rel;
            _materials[i].mu_s_si = mu_s_SI;
            _materials[i].i_mu_s_si = 1.0 / mu_s_SI;
            _materials[i].k_latt = 0.0; //::mp::material[i].Klatt_SI / mu_s_SI;
            // Divide anisotropy energy constants by mus_i to have it in units of field [T]
            _materials[i].sh2 = ku2 * _materials[i].i_mu_s_si; // J/T
            _materials[i].sh4 = ku4 * _materials[i].i_mu_s_si;
            _materials[i].sh6 = ku6 * _materials[i].i_mu_s_si;
            _materials[i].kc4 = kc4 * _materials[i].i_mu_s_si;
            _materials[i].anisotropy_unit_x = ku_vector[0];
            _materials[i].anisotropy_unit_y = ku_vector[1];
            _materials[i].anisotropy_unit_z = ku_vector[2];
            _materials[i].applied_field_strength = ::mp::material[i].applied_field_strength;
            _materials[i].applied_field_unit_x = ::mp::material[i].applied_field_unit_vector[0];
            _materials[i].applied_field_unit_y = ::mp::material[i].applied_field_unit_vector[1];
            _materials[i].applied_field_unit_z = ::mp::material[i].applied_field_unit_vector[2];
            _materials[i].temperature = ::mp::material[i].temperature;
            _materials[i].temperature_rescaling_alpha = ::mp::material[i].temperature_rescaling_alpha;
            _materials[i].temperature_rescaling_Tc = ::mp::material[i].temperature_rescaling_Tc;
            _materials[i].H_th_sigma = ::mp::material[i].H_th_sigma;
         }

         // std::vector<double> ks_tensor = ::anisotropy::get_ku_vector(i);

         /*
          * Allocate memory and send information about the materials
          */
         hipMalloc((void **)&cu::mp::d_material_params, num_mats * sizeof(material_parameters_t));
         hipMemcpy(cu::mp::d_material_params, _materials.data(), num_mats * sizeof(material_parameters_t), hipMemcpyHostToDevice);

         return true;
      }

      bool __initialize_topology()
      {

         // Transfer the row ptrs and col indices to the device
         std::vector<int> limits_h(::atoms::num_atoms + 1, 0);
         for (int atom = 0; atom < ::atoms::num_atoms; atom++)
            limits_h[atom + 1] = ::atoms::neighbour_list_end_index[atom] + 1;

         hipMalloc((void **)&cu::atoms::d_limits, (::atoms::num_atoms + 1) * sizeof(int));
         hipMalloc((void **)&cu::atoms::d_neighbours, ::atoms::neighbour_list_array.size() * sizeof(int));

         hipMemcpy(cu::atoms::d_limits, limits_h.data(), (::atoms::num_atoms + 1) * sizeof(int), hipMemcpyHostToDevice);
         hipMemcpy(cu::atoms::d_neighbours, ::atoms::neighbour_list_array.data(), ::atoms::neighbour_list_array.size() * sizeof(int), hipMemcpyHostToDevice);

         return true;
      }

      bool __initialize_curand()
      {
         hipError_t malloc_error = hipMalloc(
             (void **)&cu::d_rand_state,
             cu::grid_size * cu::block_size * sizeof(hiprandState));

         if (malloc_error != hipSuccess)
         {
            std::cout << "CUDA malloc error in __initialize_curand: " << hipGetErrorString(malloc_error) << std::endl;
            return false;
         }

         check_cuda_errors(__FILE__, __LINE__);

         cu::init_rng<<<cu::grid_size, cu::block_size>>>(
             cu::d_rand_state, ::mtrandom::integration_seed);

         hipError_t kernel_error = hipGetLastError();
         if (kernel_error != hipSuccess)
         {
            std::cout << "WARNING: CUDA kernel launch error in init_rng: " << hipGetErrorString(kernel_error) << std::endl;
            std::cout << "WARNING: Continuing anyway as Monte Carlo has its own CURAND setup" << std::endl;
            // Don't return false here, as Monte Carlo has its own CURAND generator
         }

         hipError_t sync_error = hipDeviceSynchronize();
         if (sync_error != hipSuccess)
         {
            std::cout << "WARNING: CUDA sync error after init_rng: " << hipGetErrorString(sync_error) << std::endl;
            std::cout << "WARNING: Continuing anyway as Monte Carlo has its own CURAND setup" << std::endl;
            // Clear the error state
            hipGetLastError();
         }

         check_cuda_errors(__FILE__, __LINE__);

         return true; // Return true to allow initialization to continue
      }

      bool __initialize_stats()
      {

         return true;
      }

      /**
       * Inits the random number generator states in the device, one per thread
       */
      __global__ void init_rng(hiprandState *states, int seed)
      {
         int tid = blockIdx.x * blockDim.x + threadIdx.x;
         // Bounds check to prevent out-of-bounds access
         if (tid < gridDim.x * blockDim.x)
         {
            hiprand_init(seed, tid, 0, &states[tid]);
         }
      }
   }

#endif

} // end of namespace vcuda
