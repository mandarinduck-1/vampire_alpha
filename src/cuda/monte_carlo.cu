#include "hip/hip_runtime.h"

#include "atoms.hpp"
#include "exchange.hpp"
#include "vio.hpp"
#include "cuda_utils.hpp"
#include "internal.hpp"
#include "data.hpp"
#include "material.hpp"
#include "random.hpp"
#include "cuda.hpp"
#include "cells.hpp" // 添加 cells 头文件

#include "spin_fields.hpp"

#include "montecarlo.hpp"
#include "monte_carlo.hpp"

#include <thrust/device_vector.h>

#include "../src/montecarlo/internal.hpp"

#include <vector>

namespace vcuda
{

    //--------------------------------------------------------------------------
    // Function to perform a single Monte Carlo step
    //--------------------------------------------------------------------------
    void mc_step()
    {

#ifdef CUDA
        // check for cuda initialization, and initialize if necessary
        if (!internal::mc::initialised)
            internal::mc::initialise();
        // perform a single monte carlo step
        internal::mc::__mc_step();

#endif

        return;
    }

    namespace internal
    {
        namespace mc
        {

            bool initialised(false);

            // Number of sublattices
            int M;

            cu_real_t step_size;

            // Array to store the colour of each atom
            std::vector<int> atom_colour;

            // Create a set of lists giving which atoms have which colours
            std::vector<std::vector<int>> colour_list;

            // Index starting point in continuous array
            std::vector<int> sl_start;

            // device array for atoms in (sub lattice)
            int *d_sl_atoms;

            int *d_accepted;

            int64_t seed;
            hiprandGenerator_t gen;

            // Arrays for random spin vector and acceptance probability
            cu_real_t *d_rand_spin;
            cu_real_t *d_rand_accept;

            int colour_split()
            {

                std::cout << "Starting graph colouring" << std::endl;

                int N = ::atoms::num_atoms;

                // Array to store the colour of each atom
                // All are initially uncoloured (=-1)
                std::vector<int> atom_colour;
                atom_colour.assign(N, -1);

                // initial number of colours
                M = 2;

                int Ncoloured = 0;

                // Starting atom
                int iatom = 0;

                // Array to store the number of each colour that nbr the atoms
                std::vector<int> nbr_colours(M);

                do
                {

                    // std::cout << Ncoloured << "  " << iatom << "  " << atom_colour[iatom] << "  ";
                    //  temporay constants for loop start and end indices
                    const int start = ::atoms::neighbour_list_start_index[iatom];
                    const int end = ::atoms::neighbour_list_end_index[iatom] + 1;

                    nbr_colours.assign(M, 0);
                    // loop over all neighbours and count how many of each colour
                    for (int nn = start; nn < end; ++nn)
                    {
                        const int atomj = ::atoms::neighbour_list_array[nn]; // get neighbouring atom number
                        int ac = atom_colour[atomj];
                        if (ac >= 0)
                            nbr_colours[ac]++;
                    }

                    // Go backwards through the number of nbring colours
                    // This atoms colour will be the lowest zero colour
                    int lowest = M;
                    for (int i = M - 1; i >= 0; i--)
                        if (nbr_colours[i] == 0)
                            lowest = i;

                    // If all there is no available colour increase the total and give the atom the new colour
                    if (lowest == M)
                        M++;

                    atom_colour[iatom] = lowest;
                    Ncoloured++;
                    // std::cout << "lowest = " << lowest << std::endl;

                    // Select the next atom from the next uncoloured neighbour
                    int next_atom = -1;
                    // loop over all neighbours
                    for (int nn = start; nn < end; ++nn)
                    {
                        const int atomj = ::atoms::neighbour_list_array[nn]; // get neighbouring atom number
                        int ac = atom_colour[atomj];
                        // find the lowest uncoloured nbr
                        if (ac < 0)
                        {
                            next_atom = atomj;
                            break;
                        }
                    }

                    // if all neighbours are coloured select from list
                    if (next_atom == -1)
                    {
                        for (int i = 0; i < N; i++)
                        {
                            int ac = atom_colour[i];
                            // find the lowest uncoloured nbr
                            if (ac < 0)
                            {
                                next_atom = i;
                                break;
                            }
                        }
                    }

                    // Move onto the next selected atom
                    iatom = next_atom;

                    // If we can't find another uncoloured atom, end the do while
                    if (next_atom == -1)
                    {
                        std::cout << "No more uncoloured atoms. Fraction coloured = " << Ncoloured << " / " << N << std::endl;
                        break;
                    }

                } while (Ncoloured < N);

                // Create a set of lists giving which atoms have which colours
                colour_list.resize(M);

                for (int i = 0; i < N; i++)
                {
                    int ac = atom_colour[i];
                    colour_list[ac].push_back(i);
                }

                for (int i = 0; i < M; i++)
                {
                    std::cout << "Colour: " << i << ", natoms = " << colour_list[i].size() << std::endl;
                }

                return 0;
            }

            int initialise()
            {

                hiprandStatus_t curand_status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
                if (curand_status != HIPRAND_STATUS_SUCCESS)
                {
                    printf("CURAND create generator error: %d\n", curand_status);
                    exit(-1);
                }

                curand_status = hiprandSetPseudoRandomGeneratorSeed(gen, mtrandom::integration_seed);
                if (curand_status != HIPRAND_STATUS_SUCCESS)
                {
                    printf("CURAND set seed error: %d\n", curand_status);
                    exit(-1);
                }

                // Check for CUDA errors after CURAND operations
                hipError_t curand_sync_error = hipDeviceSynchronize();
                if (curand_sync_error != hipSuccess)
                {
                    printf("CUDA error after CURAND setup at %s:%i: %s\n", __FILE__, __LINE__, hipGetErrorString(curand_sync_error));
                    exit(-1);
                }

                hipError_t malloc_error = hipMalloc((void **)&d_rand_spin, 3 * ::atoms::num_atoms * sizeof(cu_real_t));
                if (malloc_error != hipSuccess)
                {
                    printf("CUDA malloc error for d_rand_spin: %s\n", hipGetErrorString(malloc_error));
                    exit(-1);
                }

                malloc_error = hipMalloc((void **)&d_rand_accept, ::atoms::num_atoms * sizeof(cu_real_t));
                if (malloc_error != hipSuccess)
                {
                    printf("CUDA malloc error for d_rand_accept: %s\n", hipGetErrorString(malloc_error));
                    exit(-1);
                }

                malloc_error = hipMalloc((void **)&d_sl_atoms, ::atoms::num_atoms * sizeof(int));
                if (malloc_error != hipSuccess)
                {
                    printf("CUDA malloc error for d_sl_atoms: %s\n", hipGetErrorString(malloc_error));
                    exit(-1);
                }
                malloc_error = hipMalloc((void **)&d_accepted, ::atoms::num_atoms * sizeof(int));
                if (malloc_error != hipSuccess)
                {
                    printf("CUDA malloc error for d_accepted: %s\n", hipGetErrorString(malloc_error));
                    exit(-1);
                }

                // Check for CUDA errors after all memory allocations
                hipError_t malloc_sync_error = hipDeviceSynchronize();
                if (malloc_sync_error != hipSuccess)
                {
                    printf("CUDA error after memory allocations at %s:%i: %s\n", __FILE__, __LINE__, hipGetErrorString(malloc_sync_error));
                    exit(-1);
                }

                colour_split();

                // Check for CUDA errors after colour_split
                hipError_t colour_sync_error = hipDeviceSynchronize();
                if (colour_sync_error != hipSuccess)
                {
                    printf("CUDA error after colour_split at %s:%i: %s\n", __FILE__, __LINE__, hipGetErrorString(colour_sync_error));
                    exit(-1);
                }

                std::vector<int> h_sl_atoms;
                h_sl_atoms.resize(::atoms::num_atoms);
                int idx = 0;
                for (int i = 0; i < M; i++)
                {
                    sl_start.push_back(idx);
                    for (int j = 0; j < colour_list[i].size(); j++)
                    {
                        h_sl_atoms[idx] = colour_list[i][j];
                        idx++;
                    }
                }

                hipError_t copy_error = hipMemcpy(d_sl_atoms, h_sl_atoms.data(), ::atoms::num_atoms * sizeof(int), hipMemcpyHostToDevice);
                if (copy_error != hipSuccess)
                {
                    printf("CUDA memcpy error: %s\n", hipGetErrorString(copy_error));
                    exit(-1);
                }

                // Check for CUDA errors immediately after memcpy
                hipError_t sync_error = hipDeviceSynchronize();
                if (sync_error != hipSuccess)
                {
                    printf("CUDA error after memcpy at %s:%i: %s\n", __FILE__, __LINE__, hipGetErrorString(sync_error));
                    exit(-1);
                }

                std::cout << "Trying a step..." << std::endl;

                // Use modern CUDA synchronization
                hipDeviceSynchronize();
                hipError_t error = hipGetLastError();
                if (error != hipSuccess)
                {
                    printf("CUDA error at %s:%i: %s\n", __FILE__, __LINE__, hipGetErrorString(error));
                    exit(-1);
                }

                __mc_step();
                std::cout << "Done" << std::endl;

                initialised = true;

                return 0;
            }

            void finalise()
            {

                hiprandDestroyGenerator(gen);

                hipFree(d_rand_spin);
                hipFree(d_rand_accept);

                hipFree(d_sl_atoms);
                hipFree(d_accepted);
            }

            __global__ void monte_carlo_sublattice_step(
                const int sl_start,
                int *sl_atoms,
                int *material,
                vcuda::internal::material_parameters_t *material_params,
                cu_real_t *rand_spin,
                cu_real_t *rand_accept,
                int *accepted,
                cu_real_t *spin3n,
                cu_real_t *x_ext_field, cu_real_t *y_ext_field, cu_real_t *z_ext_field,
                cu_real_t *local_field_x, cu_real_t *local_field_y, cu_real_t *local_field_z, // 添加局部场参数
                int *csr_rows, int *csr_cols, cu_real_t *vals,
                const cu_real_t step_size, const cu_real_t global_temperature, const int N, const int Natoms,
                ::montecarlo::algorithm_t algorithm, cu_real_t adaptive_sigma)
            {

                // Loop over blocks for large systems > ~100k spins
                for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
                     i < N;
                     i += blockDim.x * gridDim.x)
                {

                    // load an atom from the current sublattice
                    int atom = sl_atoms[i + sl_start];

                    // Get material of atom
                    int mid = material[atom];

                    // Load parameters to local variables from memory
                    cu::material_parameters_t mat = material_params[mid];

                    // material dependent temperature rescaling
                    cu_real_t alpha = mat.temperature_rescaling_alpha;
                    cu_real_t Tc = mat.temperature_rescaling_Tc;
#ifdef CUDA_DP
                    cu_real_t rescaled_temperature = global_temperature < Tc ? Tc * pow(global_temperature / Tc, alpha) : global_temperature;
#else
                    cu_real_t rescaled_temperature = global_temperature < Tc ? Tc * __powf(global_temperature / Tc, alpha) : global_temperature;
#endif
                    cu_real_t rescaled_material_kBTBohr = 9.27400915e-24 / (rescaled_temperature * 1.3806503e-23);

#ifdef CUDA_DP
                    cu_real_t sigma = rescaled_temperature < 1.0 ? 0.02 : pow(1.0 / rescaled_material_kBTBohr, 0.2) * 0.08;
#else
                    cu_real_t sigma = rescaled_temperature < 1.0 ? 0.02 : __powf(1.0 / rescaled_material_kBTBohr, 0.2) * 0.08;
#endif

                    // load spin direction to registers for later multiple reuse
                    cu_real_t sx = spin3n[atom];
                    cu_real_t sy = spin3n[atom + Natoms];
                    cu_real_t sz = spin3n[atom + 2 * Natoms];

                    // new spin direction
                    cu_real_t nsx, nsy, nsz;

                    // run chosen move type
                    // Select algorithm using case statement
                    switch (algorithm)
                    {

                    case ::montecarlo::adaptive:
                    {
                        // TODO add in adaptive_sigma
                        nsx = sx + rand_spin[atom] * adaptive_sigma;
                        nsy = sy + rand_spin[atom + N] * adaptive_sigma;
                        nsz = sz + rand_spin[atom + 2 * N] * adaptive_sigma;

// find length using appropriate device sqrt function
#ifdef CUDA_DP
                        double mod_s = 1.0 / __dsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#else
                        float mod_s = 1.0 / __frsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#endif

                        nsx *= mod_s;
                        nsy *= mod_s;
                        nsz *= mod_s;
                        break;
                    }
                    case ::montecarlo::spin_flip:
                        nsx = -sx;
                        nsy = -sy;
                        nsz = -sz;
                        break;

                    case ::montecarlo::uniform:
                    {
                        nsx = rand_spin[atom];
                        nsy = rand_spin[atom + N];
                        nsz = rand_spin[atom + 2 * N];

// find length using appropriate device sqrt function
#ifdef CUDA_DP
                        double mod_s = 1.0 / __dsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#else
                        float mod_s = 1.0 / __frsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#endif

                        nsx *= mod_s;
                        nsy *= mod_s;
                        nsz *= mod_s;
                        break;
                    }

                    case ::montecarlo::angle:
                    {

                        nsx = sx + rand_spin[atom] * sigma;
                        nsy = sy + rand_spin[atom + N] * sigma;
                        nsz = sz + rand_spin[atom + 2 * N] * sigma;

// find length using appropriate device sqrt function
#ifdef CUDA_DP
                        double mod_s = 1.0 / __dsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#else
                        float mod_s = 1.0 / __frsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#endif

                        nsx *= mod_s;
                        nsy *= mod_s;
                        nsz *= mod_s;
                        break;
                    }

                    case ::montecarlo::hinzke_nowak:
                    {

                        // const int pick_move=int(3.0*mtrandom::grnd());
                        // mtrandom::grnd() is a random number in the half open interval [1,0)
                        int pick_move = 1;
                        switch (pick_move)
                        {
                        case 0: // spin flip
                            nsx = -sx;
                            nsy = -sy;
                            nsz = -sz;
                            break;
                        case 1: // uniform
                        {
                            nsx = rand_spin[atom];
                            nsy = rand_spin[atom + N];
                            nsz = rand_spin[atom + 2 * N];

// find length using appropriate device sqrt function
#ifdef CUDA_DP
                            double mod_s = 1.0 / __dsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#else
                            float mod_s = 1.0 / __frsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#endif

                            nsx *= mod_s;
                            nsy *= mod_s;
                            nsz *= mod_s;
                            break;
                        }

                        case 2: // angle
                        {

                            nsx = sx + rand_spin[atom] * sigma;
                            nsy = sy + rand_spin[atom + N] * sigma;
                            nsz = sz + rand_spin[atom + 2 * N] * sigma;

// find length using appropriate device sqrt function
#ifdef CUDA_DP
                            double mod_s = 1.0 / __dsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#else
                            float mod_s = 1.0 / __frsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#endif

                            nsx *= mod_s;
                            nsy *= mod_s;
                            nsz *= mod_s;
                            break;
                        }
                        default: // angle
                        {

                            nsx = sx + rand_spin[atom] * sigma;
                            nsy = sy + rand_spin[atom + N] * sigma;
                            nsz = sz + rand_spin[atom + 2 * N] * sigma;

// find length using appropriate device sqrt function
#ifdef CUDA_DP
                            double mod_s = 1.0 / __dsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#else
                            float mod_s = 1.0 / __frsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#endif

                            nsx *= mod_s;
                            nsy *= mod_s;
                            nsz *= mod_s;
                            break;
                        }
                        }
                        break;
                    }
                    default: // adaptive
                    {

                        // TODO add in adaptive_sigma
                        nsx = sx + rand_spin[atom] * adaptive_sigma;
                        nsy = sy + rand_spin[atom + N] * adaptive_sigma;
                        nsz = sz + rand_spin[atom + 2 * N] * adaptive_sigma;

// find length using appropriate device sqrt function
#ifdef CUDA_DP
                        double mod_s = 1.0 / __dsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#else
                        float mod_s = 1.0 / __frsqrt_rn(nsx * nsx + nsy * nsy + nsz * nsz);
#endif

                        nsx *= mod_s;
                        nsy *= mod_s;
                        nsz *= mod_s;
                        break;
                    }
                    }

                    // Calculate current energy
                    cu_real_t Eold = ::vcuda::internal::uniaxial_anisotropy_energy(mat, sx, sy, sz);
                    cu_real_t Enew = ::vcuda::internal::uniaxial_anisotropy_energy(mat, nsx, nsy, nsz);

                    cu_real_t hx = ::vcuda::internal::exchange::exchange_field_component(csr_cols, csr_rows, vals, spin3n, atom);
                    cu_real_t hy = ::vcuda::internal::exchange::exchange_field_component(csr_cols, csr_rows, vals, spin3n, atom + Natoms);
                    cu_real_t hz = ::vcuda::internal::exchange::exchange_field_component(csr_cols, csr_rows, vals, spin3n, atom + 2 * Natoms);

                    // ======唐愈涵加的目的是实现局部场======
                    // 添加局部场到交换场
                    hx += local_field_x[atom];
                    hy += local_field_y[atom];
                    hz += local_field_z[atom];
                    // ============================

                    cu_real_t dEx = -mat.mu_s_si * ((nsx - sx) * hx + (nsy - sy) * hy + (nsz - sz) * hz);

                    cu_real_t dE = (Enew - Eold + dEx) / (1.38064852e-23 * global_temperature);

                    cu_real_t r_accept = rand_accept[atom];

#ifdef CUDA_DP
                    if (r_accept < exp(-dE))
                    {
#else
                    if (r_accept < __expf(-dE))
                    {
#endif
                        spin3n[atom] = nsx;
                        spin3n[atom + Natoms] = nsy;
                        spin3n[atom + 2 * Natoms] = nsz;
                        accepted[i + sl_start] = 1;
                    }
                    // x_ext_field[atom] = rand_accept[atom];
                    // y_ext_field[atom] = rand_spin[atom];
                    // z_ext_field[atom] = rand_spin[atom+Natoms];
                    // x_ext_field[atom] = hx;
                    // y_ext_field[atom] = hy;
                    // z_ext_field[atom] = hz;
                }
            }

            void __mc_step()
            {
                // Check for cuda errors in file, line
                check_cuda_errors(__FILE__, __LINE__);

                // Load separate spin vectors into single array
                // hipMemcpy(cu::exchange::d_spin3n, 				            cu::atoms::d_x_spin, ::atoms::num_atoms * sizeof(cu_real_t), hipMemcpyDeviceToDevice);
                // hipMemcpy(cu::exchange::d_spin3n + ::atoms::num_atoms, 	cu::atoms::d_y_spin, ::atoms::num_atoms * sizeof(cu_real_t), hipMemcpyDeviceToDevice);
                // hipMemcpy(cu::exchange::d_spin3n + 2 * ::atoms::num_atoms, cu::atoms::d_z_spin, ::atoms::num_atoms * sizeof(cu_real_t), hipMemcpyDeviceToDevice);

                // generate 3 random doubles per atom for the trial spin and 1 for the acceptance
#ifdef CUDA_DP
                hiprandStatus_t status_normal = hiprandGenerateNormalDouble(gen, d_rand_spin, 3 * ::atoms::num_atoms, 0.0, 1.0);
                if (status_normal != HIPRAND_STATUS_SUCCESS)
                {
                    printf("CURAND error generating normal doubles: %d\n", status_normal);
                    exit(-1);
                }

                hiprandStatus_t status_uniform = hiprandGenerateUniformDouble(gen, d_rand_accept, ::atoms::num_atoms);
                if (status_uniform != HIPRAND_STATUS_SUCCESS)
                {
                    printf("CURAND error generating uniform doubles: %d\n", status_uniform);
                    exit(-1);
                }
#else
                hiprandStatus_t status_normal = hiprandGenerateNormal(gen, d_rand_spin, 3 * ::atoms::num_atoms, 0.0, 1.0);
                if (status_normal != HIPRAND_STATUS_SUCCESS)
                {
                    printf("CURAND error generating normal floats: %d\n", status_normal);
                    exit(-1);
                }
                hiprandStatus_t status_uniform = hiprandGenerateUniform(gen, d_rand_accept, ::atoms::num_atoms);
                if (status_uniform != HIPRAND_STATUS_SUCCESS)
                {
                    printf("CURAND error generating uniform floats: %d\n", status_uniform);
                    exit(-1);
                }
#endif

                hipMemset(d_accepted, 0, ::atoms::num_atoms * sizeof(int));

                // Calculate external fields (fixed for integration step)
                // cu::update_external_fields ();

                // Iterate over all the sublattices
                for (int i = 0; i < M; i++)
                {
                    if (colour_list[i].size() > 0)
                    { // Only launch if there are atoms in this sublattice
                        monte_carlo_sublattice_step<<<cu::grid_size, cu::block_size>>>(
                            sl_start[i],
                            d_sl_atoms,
                            ::cu::atoms::d_materials, cu::mp::d_material_params,
                            d_rand_spin, d_rand_accept,
                            d_accepted,
                            ::cu::atoms::d_spin,
                            ::cu::d_x_external_field, ::cu::d_y_external_field, ::cu::d_z_external_field,
                            ::cu::local_field_x, ::cu::local_field_y, ::cu::local_field_z, // 添加局部场参数
                            ::vcuda::internal::exchange::d_csr_rows, ::vcuda::internal::exchange::d_coo_cols, ::vcuda::internal::exchange::d_coo_vals,
                            step_size, sim::temperature, colour_list[i].size(), ::atoms::num_atoms,
                            ::montecarlo::algorithm, (cu_real_t)::montecarlo::internal::adaptive_sigma);

                        // Check for kernel launch errors
                        hipError_t kernel_error = hipGetLastError();
                        if (kernel_error != hipSuccess)
                        {
                            printf("CUDA kernel launch error for sublattice %d: %s\n", i, hipGetErrorString(kernel_error));
                            exit(-1);
                        }
                    }
                    else
                    {
                        // Skip empty sublattice
                    }
                }

                // hipMemcpy(cu::atoms::d_x_spin, cu::exchange::d_spin3n, 				            ::atoms::num_atoms * sizeof(cu_real_t), hipMemcpyDeviceToDevice);
                // hipMemcpy(cu::atoms::d_y_spin, cu::exchange::d_spin3n + ::atoms::num_atoms, 		::atoms::num_atoms * sizeof(cu_real_t), hipMemcpyDeviceToDevice);
                // hipMemcpy(cu::atoms::d_z_spin, cu::exchange::d_spin3n + 2 * ::atoms::num_atoms, 	::atoms::num_atoms * sizeof(cu_real_t), hipMemcpyDeviceToDevice);

                // wrap raw pointer with a device_ptr
                thrust::device_ptr<int> dev_ptr(d_accepted);
                // total number of accepted moves
                int accepted_moves = thrust::reduce(dev_ptr, dev_ptr + ::atoms::num_atoms);
                int rejected_moves = ::atoms::num_atoms - accepted_moves;

                // calculate new adaptive step sigma angle
                if (::montecarlo::algorithm == ::montecarlo::adaptive)
                {
                    const cu_real_t last_rejection_rate = rejected_moves / ::atoms::num_atoms;
                    const cu_real_t factor = 0.5 / last_rejection_rate;
                    ::montecarlo::internal::adaptive_sigma *= factor;
                    // check for excessive range (too small angles takes too long to grow, too large doesn't improve performance) and truncate
                    if (::montecarlo::internal::adaptive_sigma > 60.0 || ::montecarlo::internal::adaptive_sigma < 1e-5)
                        montecarlo::internal::adaptive_sigma = 60.0;
                }

                // Save statistics to sim namespace variable
                sim::mc_statistics_moves += ::atoms::num_atoms;
                sim::mc_statistics_reject += rejected_moves;

                vcuda::transfer_spin_positions_from_gpu_to_cpu();
                // Output debugging
                // std::vector<double> hx, hy, hz;
                // hx.resize(::atoms::num_atoms);
                // hy.resize(::atoms::num_atoms);
                // hz.resize(::atoms::num_atoms);
                // hipMemcpy(hx.data(), cu::d_x_external_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
                // hipMemcpy(hy.data(), cu::d_y_external_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
                // hipMemcpy(hz.data(), cu::d_z_external_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);

                // std::cerr << hx[0] << "  " << hy[0] << "  " << hz[0] << std::endl;
            }
        }
    }
}
